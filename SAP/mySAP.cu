#include "hip/hip_runtime.h"
#include "mySAP.h"
#include <thrust/transform_reduce.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }

/* Cuda Function */
// myFindSweepDirection's kernel
struct Obj2Diff{
	const int index, num;
	const float mean;
	Obj2Diff(const int index, const int num, const float mean): index(index), num(num), mean(mean) {}
	__host__ __device__
	float operator()(const Object &x) const{
		return ((x.pos[index] - mean)*(x.pos[index] - mean)) / num;
	}
};

struct Obj2Float{
	const int index, num;
	Obj2Float(const int index, const int num): index(index), num(num) {}
	__host__ __device__
	float operator()(const Object &x) const{
		return x.pos[index] / num;
	}
};

struct myFloatAdd{
	__host__ __device__
	float operator()(const float& a, const float& b){
		return a + b;
	}
};

// mySort's kernel
struct myCompare{
	const int index;
	myCompare(const int index): index(index) {}
	__host__ __device__
	bool operator()(const Object& a, const Object& b){
		return (a.pos[index] - a.r) < (b.pos[index] - b.r);
	}
};

// myMoveObject's kernel
__global__ void CudaMoveObject( Object *cuObj, int N, int Boundary, float FT){
	static const float g = -98.0665;	// acceleration of gravity
	static const float coef_rest = 0.9;	// coefficient of restitution
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= N) return;

	float pos[3];
	pos[0] = cuObj[id].pos[0] + ( cuObj[id].v[0]*FT );
	pos[1] = cuObj[id].pos[1] + ( cuObj[id].v[1]*FT + g*FT*FT/2 );
	pos[2] = cuObj[id].pos[2] + ( cuObj[id].v[2]*FT );

	if (pos[0] - cuObj[id].r <= 0 || pos[0] + cuObj[id].r >= Boundary){
		cuObj[id].pos[0] = (pos[0]-cuObj[id].r<=0)? cuObj[id].r: Boundary-cuObj[id].r;
		cuObj[id].pos[1] = pos[1];
		cuObj[id].pos[2] = pos[2];

		cuObj[id].v[0] = (cuObj[id].v[0] * -1);
	}
	else if (pos[2] - cuObj[id].r <= 0 || pos[2] + cuObj[id].r >= Boundary){
		cuObj[id].pos[0] = pos[0];
		cuObj[id].pos[1] = pos[1];
		cuObj[id].pos[2] = (pos[2]-cuObj[id].r<=0)? cuObj[id].r: Boundary-cuObj[id].r;

		cuObj[id].v[2] = (cuObj[id].v[2] * -1);
	}
	else if (pos[1] - cuObj[id].r <= 0){
		cuObj[id].pos[0] = pos[0];
		cuObj[id].pos[1] = cuObj[id].r;
		cuObj[id].pos[2] = pos[2];

		cuObj[id].v[1] = (cuObj[id].v[1] * -coef_rest);
	}
	else {
		cuObj[id].pos[0] = pos[0];
		cuObj[id].pos[1] = pos[1];
		cuObj[id].pos[2] = pos[2];

		cuObj[id].v[1] += g*FT;
	}
	// Set FT = 0 to use this function as reset function.
	cuObj[id].isCollision = 0;
}

// mySplitWorkspace's kernel
struct myCompareByGroup{
	__host__ __device__
	bool operator()(const Object& a, const Object& b){
		return a.group < b.group;
	}
};

__global__ void CudaGroup(	Object *cuObj, int *cuNWObj, int nWS,
							int *cuBound, int axis, int N, int Boundary){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= N) return;

	float l, r;
	l = cuObj[id].pos[axis] - cuObj[id].r;
	r = cuObj[id].pos[axis] + cuObj[id].r;

	float chunk = (float)Boundary / nWS;
	float _r;
	for (int i=0; i<nWS; i++){
		_r = (i + 1)*chunk + 0.001;
		if (_r >= r){
			atomicAdd( &(cuNWObj[i]), 1);
			cuObj[id].group = i;
			return;
		}
		else if (_r > l){
			atomicAdd( &(cuNWObj[nWS + i]), 1);
			cuObj[id].group = nWS + i;
			// offset 0 is Max bound, and nWS is Min bound.
			atomicMax( &(cuBound[i]), (int)(r-0.001+1));
			atomicMin( &(cuBound[i+nWS]), (int)l);
			return;
		}
	}
}

__global__ void CudaSplit(	int *cuTmp, int *cuBound, int *cuNWObj, Object *cuObj,
							int axis, int group, int nWS, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= N) return;

	if (cuObj[id].group == group) cuTmp[id] = 1;
	else {
		float r = cuObj[id].pos[axis] + cuObj[id].r;
		float l = cuObj[id].pos[axis] - cuObj[id].r;
		float _r = cuBound[group - nWS];
		float _l = cuBound[group];

		if ( (r > _l && r < _r) || (l < _r && l > _l)){
			atomicAdd( &(cuNWObj[group]), 1);
			cuTmp[id] = 1;
		}
		else{
			cuTmp[id] = 0;
		}
	}
}

__global__ void CudaWorkspace( Object **cuWS, int *cuTmp, Object *cuObj, int offset, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= N || cuTmp[id]==0) return;
	if (id != 0 && cuTmp[id] == cuTmp[id-1]) return;

	cuWS[offset + cuTmp[id]-1] = &(cuObj[id]);
}

// mySAP's kernel
#define sqr(a) (a)*(a)
__global__ void CudaSAP( Object *cuObj, int axis, int N){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id >= N) return;
    float pos[3], _pos[3], r, _r;
	pos[0] = cuObj[id].pos[0];
	pos[1] = cuObj[id].pos[1];
	pos[2] = cuObj[id].pos[2];
	r = cuObj[id].r;

    float bound = pos[axis] + r;
    int num = 0;
    for(int i = id + 1; i < N; i++){
		if (cuObj[i].pos[axis]-cuObj[i].r > bound) break;

        _pos[0] = cuObj[i].pos[0];
        _pos[1] = cuObj[i].pos[1];
        _pos[2] = cuObj[i].pos[2];
        _r = cuObj[i].r;
        float dist = sqr(_pos[0]-pos[0])+sqr(_pos[1]-pos[1])+sqr(_pos[2]-pos[2]);
        if(dist < (sqr(r + _r)))
			num++;
    }
	cuObj[id].isCollision = num;
}

// mySAP_WB's kernel
#define nWork 65536
__global__ void CudaWorkload( Object *cuWS, int *cuR, int *cuNT, int axis, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= N) return;
	int right;
	// Binary Search for Right (inclusive)
	{
		int l = id+1, r = N-1, mid;
		float value = cuWS[id].pos[axis] + cuWS[id].r, _v;
		while (l <= r){
			mid = (l + r) / 2;
			_v = cuWS[mid].pos[axis] - cuWS[mid].r;
			if (value > _v) l = mid + 1;
			else r = mid - 1;
		}
		right = r;
	}
	cuR[id] = right;
	cuNT[id] = CeilDiv(right - id, nWork);
}

__global__ void CudaSAP_Workload( Object *cuWS, int *cuR, int *cuNT, int axis, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	int index;
	// Binary Search for object index
	{
		int l = 0, r = N-1, mid;
		while (l <= r){
			mid = (l + r) / 2;
			if (id >= cuNT[mid]) l = mid + 1;
			else r = mid - 1;
		}
		index = l;
	}
	if (index >= N) return;

	int left, right;
	left = index != 0? id - cuNT[index-1]: id;
	left = left * nWork + index + 1;
	right = left + nWork > cuR[index]? cuR[index]+1: left+nWork;

    float pos[3], _pos[3], r, _r, dist;
    pos[0] = cuWS[index].pos[0];
    pos[1] = cuWS[index].pos[1];
    pos[2] = cuWS[index].pos[2];
    r = cuWS[index].r;

	int num = 0;
	for (int i=left; i<right; i++){
        _pos[0] = cuWS[i].pos[0];
        _pos[1] = cuWS[i].pos[1];
        _pos[2] = cuWS[i].pos[2];
        _r = cuWS[i].r;
        dist = sqr(_pos[0]-pos[0])+sqr(_pos[1]-pos[1])+sqr(_pos[2]-pos[2]);
        if(dist < (sqr(r + _r)))
			num++;
    }
	atomicAdd( &(cuWS[index].isCollision), num);
}

// mySAP_WS's kernel
#define nWorkWS 65536
__global__ void CudaWorkload_WS( Object **cuWS, int *cuR, int *cuNT, int axis, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= N) return;
	int right;
	// Binary Search for Right (inclusive)
	{
		int l = id+1, r = N-1, mid;
		float value = cuWS[id]->pos[axis] + cuWS[id]->r, _v;
		while (l <= r){
			mid = (l + r) / 2;
			_v = cuWS[mid]->pos[axis] - cuWS[mid]->r;
			if (value > _v) l = mid + 1;
			else r = mid - 1;
		}
		right = r;
	}
	cuR[id] = right;
	cuNT[id] = CeilDiv(right - id, nWorkWS);
}

__global__ void CudaSAP_Workload_WS( Object **cuWS, int *cuR, int *cuNT, int axis, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	int index;
	// Binary Search for object index
	{
		int l = 0, r = N-1, mid;
		while (l <= r){
			mid = (l + r) / 2;
			if (id >= cuNT[mid]) l = mid + 1;
			else r = mid - 1;
		}
		index = l;
	}
	if (index >= N) return;

	int left, right;
	left = index != 0? id - cuNT[index-1]: id;
	left = left * nWorkWS + index + 1;
	right = left + nWorkWS > cuR[index]? cuR[index]+1: left+nWorkWS;

    float pos[3], _pos[3], r, _r, dist;
    pos[0] = cuWS[index]->pos[0];
    pos[1] = cuWS[index]->pos[1];
    pos[2] = cuWS[index]->pos[2];
    r = cuWS[index]->r;

	int num = 0;
	for (int i=left; i<right; i++){
        _pos[0] = cuWS[i]->pos[0];
        _pos[1] = cuWS[i]->pos[1];
        _pos[2] = cuWS[i]->pos[2];
        _r = cuWS[i]->r;
        dist = sqr(_pos[0]-pos[0])+sqr(_pos[1]-pos[1])+sqr(_pos[2]-pos[2]);
        if(dist < (sqr(r + _r)))
			num++;
    }
	atomicAdd( &(cuWS[index]->isCollision), num);
}

// myPrint's kernel
__global__ void CudaDrawObject(int *cuTmp, Object *cuObj, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= N) return;
	if ( cuObj[id].isDraw) cuTmp[id] = 1;
	else cuTmp[id] = 0;
}

__global__ void CudaPrintObject(FileObject *cuFileObj, Object *cuObj, int *cuTmp, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= N || cuTmp[id] == 0) return;

	if (id == 0 || cuTmp[id] != cuTmp[id-1]){
		int _id = cuTmp[id] - 1;
		cuFileObj[_id].pos[0] = cuObj[id].pos[0];
		cuFileObj[_id].pos[1] = cuObj[id].pos[1];
		cuFileObj[_id].pos[2] = cuObj[id].pos[2];
		cuFileObj[_id].r = cuObj[id].r;
		cuFileObj[_id].isCollision = cuObj[id].isCollision;
	}
}

/**/

#define BlockSize 256

void myFindSweepDirection( Object *cuObj, int *SweepDir, int N){
	float mean, var, maxVar = -1, secVar = -1;
	int coor, coor2;
	for (int i=0; i<3; i++){
		mean = thrust::transform_reduce(thrust::device, cuObj, cuObj+N,
											Obj2Float(i, N), 0.0f, myFloatAdd());
		var = thrust::transform_reduce(thrust::device, cuObj, cuObj+N,
											Obj2Diff(i, N, mean), 0.0f, myFloatAdd());
		if (var > maxVar){
			secVar = maxVar;
			coor2 = coor;

			maxVar = var;
			coor = i;
		}
		else if (var > secVar){
			secVar = var;
			coor2 = i;
		}
	}
	SweepDir[0] = coor;
	SweepDir[1] = coor2;
}

void mySort( Object *cuObj, int *SweepDir, int N){
	thrust::sort(thrust::device, cuObj, cuObj+N, myCompare(SweepDir[0]));
}

void myWS(	Object **cuWS, int *cuNWObj, int nWS, Object *cuObj,
			int *cuTmp, int *SweepDir, int N, int Boundary)
{
	if (nWS <= 1) return;

	hipMemset( cuNWObj, 0, sizeof(int)*nWS*2);
	// Hide the O_bd boundary info. at cuNWObj tail.
	int *cuBound = cuNWObj + nWS*2;
	hipMemset( cuBound, 0, sizeof(int)*nWS);
	hipMemset( cuBound+nWS, 1, sizeof(int)*nWS);

	dim3 grid(CeilDiv(N, BlockSize)), block(BlockSize);
	CudaGroup<<< grid, block >>>(	cuObj, cuNWObj, nWS, cuBound,
									SweepDir[1], N, Boundary);

	for (int i=0; i<nWS-1; i++){
		int group = i + nWS;
		CudaSplit<<< grid, block >>>( cuTmp, cuBound, cuNWObj, cuObj,
										SweepDir[1], group, nWS, N);
		thrust::inclusive_scan(thrust::device, cuTmp, cuTmp+N, cuTmp);
		CudaWorkspace<<< grid, block >>>( cuWS, cuTmp, cuObj, i*N, N);
	}
}

void mySAP( Object *cuObj, int *SweepDir, int N){
	dim3 grid(CeilDiv(N, BlockSize)), block(BlockSize);
	CudaSAP<<< grid, block >>>( cuObj, SweepDir[0], N);
}

void mySAP_WB( Object *cuObj, int *cuR, int *cuNT, int *SweepDir, int N){
	dim3 grid(CeilDiv(N, BlockSize)), block(BlockSize);
	CudaWorkload<<< grid, block >>>( cuObj, cuR, cuNT, SweepDir[0], N);

	thrust::inclusive_scan(thrust::device, cuNT, cuNT+N, cuNT);

	int nThreads;
	hipMemcpy( &nThreads, cuNT+N-1, sizeof(int), hipMemcpyDeviceToHost);
	
	dim3 grid2(CeilDiv(nThreads, 1024)), block2(1024);
	CudaSAP_Workload<<< grid2, block2 >>>( cuObj, cuR, cuNT, SweepDir[0], N);
}

void mySAP_WS(Object *cuObj, Object **cuWS, int *cuNWObj, int nWS, int *cuR, int *cuNT, int *SweepDir, int N){
	if (nWS <= 1){
		mySAP_WB( cuObj, cuR, cuNT, SweepDir, N);
		return;
	}

	int nWObj[nWS*2];
	hipMemcpy( nWObj, cuNWObj, sizeof(int)*nWS*2, hipMemcpyDeviceToHost);

	for (int i=0; i<nWS-1; i++){
		int group = i + nWS;
		Object **_cuWS = cuWS + N*i;
		dim3 grid(CeilDiv(nWObj[group], BlockSize)), block(BlockSize);
		CudaWorkload_WS<<< grid, block >>>( _cuWS, cuR, cuNT, SweepDir[0], nWObj[group]);

		thrust::inclusive_scan(thrust::device, cuNT, cuNT+nWObj[group], cuNT);

		int nThreads;
		hipMemcpy( &nThreads, cuNT+nWObj[group]-1, sizeof(int), hipMemcpyDeviceToHost);

		dim3 grid2(CeilDiv(nThreads, 1024)), block2(1024);
		CudaSAP_Workload_WS<<< grid2, block2 >>>( _cuWS, cuR, cuNT, SweepDir[0], nWObj[group]);
	}

	thrust::stable_sort(thrust::device, cuObj, cuObj+N, myCompareByGroup());

	int offset[nWS];
	offset[0] = 0;
	for (int i=1; i<nWS; i++)
		offset[i] = offset[i-1] + nWObj[i-1];

	for (int i=0; i<nWS; i++){
		Object *_cuObj = cuObj + offset[i];
		dim3 grid(CeilDiv(nWObj[i], BlockSize)), block(BlockSize);
		CudaWorkload<<< grid, block >>>( _cuObj, cuR, cuNT, SweepDir[0], nWObj[i]);

		thrust::inclusive_scan(thrust::device, cuNT, cuNT+nWObj[i], cuNT);

		int nThreads;
		hipMemcpy( &nThreads, cuNT+nWObj[i]-1, sizeof(int), hipMemcpyDeviceToHost);

		dim3 grid2(CeilDiv(nThreads, 1024)), block2(1024);
		CudaSAP_Workload<<< grid2, block2 >>>( _cuObj, cuR, cuNT, SweepDir[0], nWObj[i]);
	}
}

void myPrint(
	Object *cuObj, FileObject *cuFileObj, FileObject *fileObj, int *cuTmp,
	int N, float frameTime, FILE *fptr
){
	dim3 grid(CeilDiv(N, BlockSize)), block(BlockSize);
	CudaDrawObject<<< grid, block >>>( cuTmp, cuObj, N);
	thrust::inclusive_scan(thrust::device, cuTmp, cuTmp+N, cuTmp);

	CudaPrintObject<<< grid, block >>>(cuFileObj, cuObj, cuTmp, N);

	int nFileObj;
	hipMemcpy( &nFileObj, cuTmp+N-1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( fileObj, cuFileObj, sizeof(FileObject)*nFileObj, hipMemcpyDeviceToHost);

	fwrite( &nFileObj, sizeof(int), 1, fptr);
	fwrite( fileObj, sizeof(FileObject), nFileObj, fptr);
	fwrite( &frameTime, sizeof(float), 1, fptr);
}

void myMoveObject( Object *cuObj, int N, int Boundary, float FT){
	dim3 grid(CeilDiv(N, BlockSize)), block(BlockSize);
	CudaMoveObject<<< grid, block >>>(cuObj, N, Boundary, FT);
}
