#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include "mySAP.h"

//#define DEBUG

int main(){
	// Define
	static const int MAXN = 10000;
	static const int Boundary = 1000;
	static const int RadiusN = 6;
	static const float Radius[] = { 1, 2, 4, 8, 16, 32};
	static const float FPS = 50;
	static const float FrameTime = (float)1 / FPS;
	static const float SimulationTime = 5;

	// Allocation
	static Object obj[MAXN];
	static Object *cuObj;
	static FileObject fileObj[MAXN];
	static FileObject *cuFileObj;
	static int SweepDir[3] = { 1, 0, 0};
	static int *cuSweepDir;

	FILE *fptr = fopen("log", "wb");

	// Preprocessing
	{
		srand(time(NULL));
		for (int i=0; i<MAXN; i++){
			obj[i].r = Radius[rand() % RadiusN];
			obj[i].pos[0] = (rand() % (int)(Boundary-2*obj[i].r)) + obj[i].r;	// X
			obj[i].pos[1] = Boundary - obj[i].r - (rand()%(Boundary/10));		// Y
			obj[i].pos[2] = (rand() % (int)(Boundary-2*obj[i].r)) + obj[i].r;	// Z
			obj[i].v[0] = (rand() % (Boundary/5+1)) - Boundary/10;
			obj[i].v[1] = 0;
			obj[i].v[2] = (rand() % (Boundary/5+1)) - Boundary/10;
			obj[i].isCollision = 0;
		}
		hipMalloc( &cuObj, sizeof(Object)*MAXN);
		hipMemcpy( cuObj, obj, sizeof(Object)*MAXN, hipMemcpyHostToDevice);

		hipMalloc( &cuFileObj, sizeof(FileObject)*MAXN);

		hipMalloc( &cuSweepDir, sizeof(int)*3);
		hipMemcpy( cuSweepDir, SweepDir, sizeof(int)*3, hipMemcpyHostToDevice);

		fwrite( &Boundary, sizeof(int), 1, fptr);
		fwrite( &MAXN, sizeof(int), 1, fptr);
	}

	#ifdef DEBUG
	{
		printf("FrameTime = %lfs\n", FrameTime);
	}
	#endif

	// Simulation
	clock_t begin = clock();
	{
		while ( ((float)(clock()-begin) / CLOCKS_PER_SEC) <= SimulationTime ){
			clock_t duration = clock();
			// SAP
			{
				myFindSweepDirection( cuObj, cuSweepDir, MAXN);
				mySort( cuObj, cuSweepDir, MAXN);
				mySAP( cuObj, cuSweepDir, MAXN);
			}
			duration = clock() - duration;

			myPrint( fptr, cuObj, cuFileObj, fileObj, MAXN, (float)duration/CLOCKS_PER_SEC);
			myMoveObject( cuObj, MAXN, Boundary, FrameTime);
		}
	}

	// Free
	{
		hipFree( cuObj);
		hipFree( cuFileObj);
		hipFree( cuSweepDir);
		fclose(fptr);
	}
	return 0;
}
